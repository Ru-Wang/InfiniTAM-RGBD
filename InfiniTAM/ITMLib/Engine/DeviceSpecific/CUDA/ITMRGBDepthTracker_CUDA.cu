#include "hip/hip_runtime.h"
#include "ITMRGBDepthTracker_CUDA.h"

#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMRGBDepthTracker.h"
#include "../../DeviceAgnostic/ITMPixelUtils.h"
#include "../../../../ORUtils/CUDADefines.h"

using namespace ITMLib::Engine;

__global__
void RGBDepthTrackerOneLevel_f_device(Vector2f* f_device,
                                      const Vector4f* locations, const Vector4f* colours, const Vector4u* rgb,
                                      Vector2i rgbImageSize, Vector4f projParams, Matrix4f M, int noTotalPoints);

__global__
void RGBDepthTrackerOneLevel_g_rt_device(float* g_device, float* h_device,
                                         const Vector4f* locations, const Vector4f* colours,
                                         const Vector4u* rgb, const float* depth,
                                         const Vector4s* gx, const Vector4s* gy,
                                         const float* gdx, const float* gdy,
                                         Vector2i rgbImageSize, Vector2i depthImageSize,
                                         Vector4f projRGBParams, Vector4f projDepthParams,
                                         Matrix4f toRGBM, Matrix4f toDepthM, int noTotalPoints);

__global__
void RGBDepthTrackerOneLevel_g_ro_device(float* g_device, float* h_device,
                                         const Vector4f* locations, const Vector4f* colours,
                                         const Vector4u* rgb, const float* depth,
                                         const Vector4s* gx, const Vector4s* gy,
                                         const float* gdx, const float* gdy,
                                         Vector2i rgbImageSize, Vector2i depthImageSize,
                                         Vector4f projRGBParams, Vector4f projDepthParams,
                                         Matrix4f toRGBM, Matrix4f toDepthM, int noTotalPoints);

ITMRGBDepthTracker_CUDA::ITMRGBDepthTracker_CUDA(Vector2i imgSize,
                                                 TrackerIterationType* trackingRegime,
                                                 int noHierarchyLevels,
                                                 const ITMLowLevelEngine* lowLevelEngine)
    : ITMRGBDepthTracker(imgSize, trackingRegime, noHierarchyLevels, lowLevelEngine, MEMORYDEVICE_CUDA) {
  const int numPara = 6;
  const int numParaSQ = 6 + 5 + 4 + 3 + 2 + 1;

  ITMSafeCall(hipMalloc((void**)&f_device, sizeof(Vector2f) * imgSize.x * imgSize.y / 128));
  ITMSafeCall(hipMalloc((void**)&g_device, sizeof(float) * numPara * (imgSize.x * imgSize.y / 128)));
  ITMSafeCall(hipMalloc((void**)&h_device, sizeof(float) * numParaSQ * (imgSize.x * imgSize.y / 128)));

  f_host = new Vector2f[imgSize.x * imgSize.y / 128];
  g_host = new float[numPara * (imgSize.x * imgSize.y / 128)];
  h_host = new float[numParaSQ * (imgSize.x * imgSize.y / 128)];
}

ITMRGBDepthTracker_CUDA::~ITMRGBDepthTracker_CUDA() {
  ITMSafeCall(hipFree(f_device));
  ITMSafeCall(hipFree(g_device));
  ITMSafeCall(hipFree(h_device));

  delete [] f_host;
  delete [] g_host;
  delete [] h_host;
}

void ITMRGBDepthTracker_CUDA::F_oneLevel(float* f, ITMPose* toRGBPose) {
  int noTotalPoints = trackingState->pointCloud->noTotalPoints;

  Vector4f projParams = view->calib->intrinsics_rgb.projectionParamsSimple.all;
  projParams.x /= 1 << levelId; projParams.y /= 1 << levelId;
  projParams.z /= 1 << levelId; projParams.w /= 1 << levelId;
  Vector2i rgbImageSize = viewHierarchy->levels[levelId]->rgb->noDims;
  Matrix4f M = toRGBPose->GetM();

  Vector4f *locations = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
  Vector4f *colours = trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
  Vector4u *rgb = viewHierarchy->levels[levelId]->rgb->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(128);
  dim3 gridSize((int)ceil(noTotalPoints * 1.0 / 128));

  memset(f_host, 0, sizeof(Vector2f) * gridSize.x);
  ITMSafeCall(hipMemset(f_device, 0, sizeof(Vector2f) * gridSize.x));
  RGBDepthTrackerOneLevel_f_device<<<gridSize, blockSize>>>(f_device, locations, colours, rgb,
                                                            rgbImageSize, projParams, M, noTotalPoints);
  ITMSafeCall(hipMemcpy(f_host, f_device, sizeof(Vector2f) * gridSize.x, hipMemcpyDeviceToHost));

  float final_f = 0;
  countedPoints_valid = 0;
  for (size_t i = 0; i < gridSize.x; ++i) {
    if (f_host[i].y > 0) {
      final_f += f_host[i].x;
      countedPoints_valid += (int)f_host[i].y;
    }
  }

  float scaleForOcclusions = 0;
  if (countedPoints_valid == 0) {
    final_f = MY_INF;
    scaleForOcclusions = 1.0;
  } else {
    scaleForOcclusions = (float)noTotalPoints / countedPoints_valid;
  }

  f[0] = final_f * scaleForOcclusions;
}

void ITMRGBDepthTracker_CUDA::G_oneLevel(float* gradient, float* hessian,
                                         ITMPose* toRGBPose, ITMPose* toDepthPose) const {
  int noTotalPoints = trackingState->pointCloud->noTotalPoints;

  Vector4f projRGBParams = view->calib->intrinsics_rgb.projectionParamsSimple.all;
  Vector4f projDepthParams = view->calib->intrinsics_d.projectionParamsSimple.all;
  projRGBParams.x /= 1 << levelId; projRGBParams.y /= 1 << levelId;
  projRGBParams.z /= 1 << levelId; projRGBParams.w /= 1 << levelId;
  projDepthParams.x /= 1 << levelId; projDepthParams.y /= 1 << levelId;
  projDepthParams.z /= 1 << levelId; projDepthParams.w /= 1 << levelId;

  Matrix4f toRGBM = toRGBPose->GetM();
  Matrix4f toDepthM = toDepthPose->GetM();

  Vector2i rgbImageSize = viewHierarchy->levels[levelId]->rgb->noDims;
  Vector2i depthImageSize = viewHierarchy->levels[levelId]->depth->noDims;

  bool rotationOnly = (iterationType == TRACKER_ITERATION_ROTATION);
  int numPara = rotationOnly ? 3 : 6;
  int numParaSQ = rotationOnly ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;

  Vector4f *locations = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
  Vector4f *colours = trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
  Vector4u *rgb = viewHierarchy->levels[levelId]->rgb->GetData(MEMORYDEVICE_CUDA);
  Vector4s *gx = viewHierarchy->levels[levelId]->gradientX_rgb->GetData(MEMORYDEVICE_CUDA);
  Vector4s *gy = viewHierarchy->levels[levelId]->gradientY_rgb->GetData(MEMORYDEVICE_CUDA);
  float *depth = viewHierarchy->levels[levelId]->depth->GetData(MEMORYDEVICE_CUDA);
  float *gdx = viewHierarchy->levels[levelId]->gradientX_depth->GetData(MEMORYDEVICE_CUDA);
  float *gdy = viewHierarchy->levels[levelId]->gradientY_depth->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(128);
  dim3 gridSize((int)ceil(noTotalPoints * 1.0 / 128));

  memset(g_host, 0, sizeof(float) * numPara * gridSize.x);
  memset(h_host, 0, sizeof(float) * numParaSQ * gridSize.x);
  ITMSafeCall(hipMemset(g_device, 0, sizeof(float) * numPara * gridSize.x));
  ITMSafeCall(hipMemset(h_device, 0, sizeof(float) * numParaSQ * gridSize.x));
  if (rotationOnly) {
    RGBDepthTrackerOneLevel_g_ro_device<<<gridSize, blockSize>>>(g_device, h_device,
                                                                 locations, colours,
                                                                 rgb, depth,
                                                                 gx, gy, gdx, gdy,
                                                                 rgbImageSize, depthImageSize,
                                                                 projRGBParams, projDepthParams,
                                                                 toRGBM, toDepthM, noTotalPoints);
  } else {
    RGBDepthTrackerOneLevel_g_rt_device<<<gridSize, blockSize>>>(g_device, h_device,
                                                                 locations, colours,
                                                                 rgb, depth,
                                                                 gx, gy, gdx, gdy,
                                                                 rgbImageSize, depthImageSize,
                                                                 projRGBParams, projDepthParams,
                                                                 toRGBM, toDepthM, noTotalPoints);
  }
  ITMSafeCall(hipMemcpy(g_host, g_device, sizeof(float) * numPara * gridSize.x, hipMemcpyDeviceToHost));
  ITMSafeCall(hipMemcpy(h_host, h_device, sizeof(float) * numParaSQ * gridSize.x, hipMemcpyDeviceToHost));

  for (size_t blockId = 1; blockId < gridSize.x; ++blockId) {
    for (int i = 0; i < numPara; ++i)
      g_host[i] += g_host[blockId * numPara + i];
    for (int i = 0; i < numParaSQ; ++i)
      h_host[i] += h_host[blockId * numParaSQ + i];
  }

  float scaleForOcclusions = (float)noTotalPoints / countedPoints_valid;
  if (countedPoints_valid == 0)
    scaleForOcclusions = 1;

  // Expand the matrice
  for (int para = 0, counter = 0; para < numPara; para++) {
    gradient[para] = g_host[para] * scaleForOcclusions;
    for (int col = 0; col <= para; col++, counter++)
      hessian[para + col * numPara] = h_host[counter] * scaleForOcclusions;
  }
  for (int row = 0; row < numPara; row++) {
    for (int col = row + 1; col < numPara; col++)
      hessian[row + col * numPara] = hessian[col + row * numPara];
  }
}

__global__
void RGBDepthTrackerOneLevel_f_device(
    Vector2f* f_device,
    const Vector4f* locations, const Vector4f* colours, const Vector4u* rgb,
    Vector2i rgbImageSize, Vector4f projParams, Matrix4f M, int noTotalPoints) {
  int locId_local = threadIdx.x;
  int locId_global = threadIdx.x + blockIdx.x * blockDim.x;

	__shared__ ORUtils::Vector2_<float> f_shared[128];

  if (locId_global < noTotalPoints) {
    float colorDiffSq = getColorDifferenceSq(locations, colours, rgb, rgbImageSize, locId_global, projParams, M);
    if (colorDiffSq >= 0) {
      f_shared[locId_local].x = colorDiffSq;
      f_shared[locId_local].y = 1;
    } else {
      f_shared[locId_local].x = 0;
      f_shared[locId_local].y = 0;
    }
  } else {
    f_shared[locId_local].x = 0;
    f_shared[locId_local].y = 0;
  }
  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
    if (locId_local < offset) {
      f_shared[locId_local].x += f_shared[offset + locId_local].x;
      f_shared[locId_local].y += f_shared[offset + locId_local].y;
    }
    __syncthreads();
  }

  if (locId_local == 0) {
    f_device[blockIdx.x].x = f_shared[locId_local].x;
    f_device[blockIdx.x].y = f_shared[locId_local].y;
  }
}

__global__
void RGBDepthTrackerOneLevel_g_rt_device(float* g_device, float* h_device,
                                         const Vector4f* locations, const Vector4f* colours,
                                         const Vector4u* rgb, const float* depth,
                                         const Vector4s* gx, const Vector4s* gy,
                                         const float* gdx, const float* gdy,
                                         Vector2i rgbImageSize, Vector2i depthImageSize,
                                         Vector4f projRGBParams, Vector4f projDepthParams,
                                         Matrix4f toRGBM, Matrix4f toDepthM, int noTotalPoints) {
  int locId_local = threadIdx.x;
  int locId_global = threadIdx.x + blockIdx.x * blockDim.x;

  const int startPara = 0;
  const int numPara = 6;
  const int numParaSQ = 6 + 5 + 4 + 3 + 2 + 1;
  __shared__ float g_h_shared[numParaSQ * 128];

  // Compute Hessian matrix first
  float localGradient[numPara];
  float localHessian[numParaSQ];
  memset(localGradient, 0, sizeof(float) * numPara);
  memset(localHessian, 0, sizeof(float) * numParaSQ);
  if (locId_global < noTotalPoints) {
    computePerPointGH_rt_RGBDepth(localGradient, localHessian,
                                  locations, colours, rgb, depth,
                                  rgbImageSize, depthImageSize, locId_global,
                                  projRGBParams, projDepthParams, toRGBM, toDepthM,
                                  gx, gy, gdx, gdy, numPara, startPara);
    for (int i = 0; i < numParaSQ; ++i) 
      g_h_shared[locId_local * numParaSQ + i] = localHessian[i];
  } else {
    memset(g_h_shared + locId_local * numPara, 0, sizeof(float) * numParaSQ);
  }
  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
    if (locId_local < offset) {
      for (int i = 0; i < numParaSQ; ++i)
        g_h_shared[locId_local * numParaSQ + i] += g_h_shared[(offset + locId_local) * numParaSQ + i];
    }
    __syncthreads();
  }

  if (locId_local == 0) {
    for (int i = 0; i < numParaSQ; ++i)
      h_device[blockIdx.x * numParaSQ + i] = g_h_shared[i];
  }
  __syncthreads();

  // Compute gradient
  if (locId_global < noTotalPoints) {
    for (int i = 0; i < numPara; ++i)
      g_h_shared[locId_local * numPara + i] = localGradient[i];
  } else {
    memset(g_h_shared + locId_local * numPara, 0, sizeof(float) * numPara);
  }
  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
    if (locId_local < offset) {
      for (int i = 0; i < numPara; ++i)
        g_h_shared[locId_local * numPara + i] += g_h_shared[(offset + locId_local) * numPara + i];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    for (int i = 0; i < numPara; ++i)
      g_device[blockIdx.x * numPara + i] = g_h_shared[i];
  }
}

__global__
void RGBDepthTrackerOneLevel_g_ro_device(float* g_device, float* h_device,
                                         const Vector4f* locations, const Vector4f* colours,
                                         const Vector4u* rgb, const float* depth,
                                         const Vector4s* gx, const Vector4s* gy,
                                         const float* gdx, const float* gdy,
                                         Vector2i rgbImageSize, Vector2i depthImageSize,
                                         Vector4f projRGBParams, Vector4f projDepthParams,
                                         Matrix4f toRGBM, Matrix4f toDepthM, int noTotalPoints) {
  int locId_local = threadIdx.x;
  int locId_global = threadIdx.x + blockIdx.x * blockDim.x;

  const int startPara = 3;
  const int numPara = 3;
  const int numParaSQ = 3 + 2 + 1;
  __shared__ float g_h_shared[numParaSQ * 128];

  // Compute Hessian matrix first
  float localGradient[numPara];
  float localHessian[numParaSQ];
  memset(localGradient, 0, sizeof(float) * numPara);
  memset(localHessian, 0, sizeof(float) * numParaSQ);
  if (locId_global < noTotalPoints) {
    computePerPointGH_rt_RGBDepth(localGradient, localHessian,
                                  locations, colours, rgb, depth,
                                  rgbImageSize, depthImageSize, locId_global,
                                  projRGBParams, projDepthParams, toRGBM, toDepthM,
                                  gx, gy, gdx, gdy, numPara, startPara);
    for (int i = 0; i < numParaSQ; ++i) 
      g_h_shared[locId_local * numParaSQ + i] = localHessian[i];
  } else {
    memset(g_h_shared + locId_local * numPara, 0, sizeof(float) * numParaSQ);
  }
  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
    if (locId_local < offset) {
      for (int i = 0; i < numParaSQ; ++i)
        g_h_shared[locId_local * numParaSQ + i] += g_h_shared[(offset + locId_local) * numParaSQ + i];
    }
    __syncthreads();
  }

  if (locId_local == 0) {
    for (int i = 0; i < numParaSQ; ++i)
      h_device[blockIdx.x * numParaSQ + i] = g_h_shared[i];
  }
  __syncthreads();

  // Compute gradient
  if (locId_global < noTotalPoints) {
    for (int i = 0; i < numPara; ++i)
      g_h_shared[locId_local * numPara + i] = localGradient[i];
  } else {
    memset(g_h_shared + locId_local * numPara, 0, sizeof(float) * numPara);
  }
  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
    if (locId_local < offset) {
      for (int i = 0; i < numPara; ++i)
        g_h_shared[locId_local * numPara + i] += g_h_shared[(offset + locId_local) * numPara + i];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    for (int i = 0; i < numPara; ++i)
      g_device[blockIdx.x * numPara + i] = g_h_shared[i];
  }
}